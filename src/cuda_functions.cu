#include <hip/hip_runtime.h>

#include <cstdio>

// #include "cuda_functions.h"

#define CONV(l, c, nb_c) \
(l) * (nb_c) + (c)

extern "C" void test() {
    int deviceCount;
    hipError_t cuda_status = hipGetDeviceCount(&deviceCount);
    int driverVersion;
    hipError_t cudaStatus2 = hipDriverGetVersion(&driverVersion);

    printf("%d", driverVersion);
    if (cudaStatus2 != hipSuccess) {
        printf("driver failed: %s\n", hipGetErrorString(cudaStatus2));
        return;
    }
    if (cuda_status != hipSuccess) {
        printf("hipGetDeviceCount failed: %s\n", hipGetErrorString(cuda_status));
        return;
    }
}

extern "C" void test2(int* image) {
    int* d_image;
    hipSetDevice(0);
    hipMalloc(&d_image, sizeof(int));
    int end = 0;
    printf("before %d\n", image[0]);

    hipError_t cuda_status = hipMemcpy(d_image, &end, sizeof(int), hipMemcpyHostToDevice);
    printf("error?: %s\n", hipGetErrorString(cuda_status));

    hipMemcpy(image, d_image, sizeof(int), hipMemcpyDeviceToHost);
    printf("after %d\n", image[0]);
}

__global__ void apply_blur_filter_kernel(int* image, int* new_image, int *end, int size, int* threshold, int width, int height) {
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    if (j >= 0 && j < height && i >= 0 && i < width) {
        new_image[j * width + i] = image[j * width + i];
        if (j >= size && j < height / 10 - size) {
            if (i >= size && i < width - size) {
                int stencil_j, stencil_k;
                int t = 0;

                for (stencil_j = -size; stencil_j <= size; stencil_j++) {
                    for (stencil_k = -size; stencil_k <= size; stencil_k++) {
                        t += image[(j + stencil_j) * width + (i + stencil_k)];
                    }
                }
                new_image[j * width + i] = t / ((2 * size + 1) * (2 * size + 1));
            }
        }

        if (j >= height * 0.9 + size && j < height - size) {
            if (i >= size && i < width - size) {
                int stencil_j, stencil_k;
                int t = 0;

                for (stencil_j = -size; stencil_j <= size; stencil_j++) {
                    for (stencil_k = -size; stencil_k <= size; stencil_k++) {
                        t += image[(j + stencil_j) * width + (i + stencil_k)];
                    }
                }

                new_image[j * width + i] = t / ((2 * size + 1) * (2 * size + 1));
            }
        }

        __syncthreads();

        if (j > 0 && j < height - 1 && i > 0 && i < width - 1) {
            float diff = new_image[j * width + i] - image[j * width + i];
            if (diff > *threshold || -diff > *threshold) {
                atomicAnd(end, 0);
            }
            image[j * width + i] = new_image[j * width + i];
        }
    }
}

extern "C" void apply_blur_filter_cuda(int* image, int threshold, int size, int width, int height) {
    int* d_image;
    int* d_new_image;
    int* d_threshold;
    int* d_end;
    int end = 0;

    hipMalloc(&d_image, width * height * sizeof(int));
    hipMalloc(&d_new_image, width * height * sizeof(int));
    hipMalloc(&d_threshold, sizeof(int));
    hipMalloc(&d_end, sizeof(int));
    hipMemcpy(d_image, image, width * height * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_threshold, &threshold, sizeof(int), hipMemcpyHostToDevice);
    dim3 block_size(32, 32);
    dim3 grid_size((width + block_size.x - 1) / block_size.x + 1, (height + block_size.y - 1) / block_size.y + 1, 1);

    do {
        end = 1;
        hipMemcpy(d_end, &end, sizeof(int), hipMemcpyHostToDevice);
        apply_blur_filter_kernel<<<grid_size, block_size>>>(d_image, d_new_image, d_end, size, d_threshold, width, height);

        hipMemcpy(image, d_image, width * height * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&end, d_end, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&threshold, d_threshold, sizeof(int), hipMemcpyDeviceToHost);

    } while (!end);
    hipFree(d_image);
    hipFree(d_new_image);
    hipFree(d_threshold);
    hipFree(d_end);
}



__global__ void sobel_filter_kernel(int* image, int* sobel, int width, int height) {
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (j > 0 && j < height - 1 && k > 0 && k < width - 1) {
        int pixel_no = image[CONV(j - 1, k - 1, width)];
        int pixel_n = image[CONV(j - 1, k, width)];
        int pixel_ne = image[CONV(j - 1, k + 1, width)];
        int pixel_so = image[CONV(j + 1, k - 1, width)];
        int pixel_s = image[CONV(j + 1, k, width)];
        int pixel_se = image[CONV(j + 1, k + 1, width)];
        int pixel_o = image[CONV(j, k - 1, width)];
        int pixel_e = image[CONV(j, k + 1, width)];
        float deltaX = -pixel_no + pixel_ne - 2 * pixel_o + 2 * pixel_e - pixel_so + pixel_se;
        float deltaY = pixel_se + 2 * pixel_s + pixel_so - pixel_ne - 2 * pixel_n - pixel_no;
        float val = sqrtf(deltaX * deltaX + deltaY * deltaY) / 4;
        if (val > 50) {
            sobel[CONV(j, k, width)] = 255;
        }
        else {
            sobel[CONV(j, k, width)] = 0;
        }
    }
    else if(((j==0 || j == height-1) && (k>=0 && k<width)) || ((k==0 || k == width-1) && (j>=0 && j<height-1))) {
        sobel[CONV(j, k, width)] = image[CONV(j, k, width)];
    }
}

extern "C" void apply_sobel_filter_cuda(int* image, int width, int height) {
    int* d_image = NULL;
    int* d_sobel = NULL;
    hipError_t cuda_error = hipMalloc((void**)&d_image, width * height * sizeof(int));
    if (cuda_error != hipSuccess) {
        fprintf(stderr, "(hipMalloc d_image): %s\n", hipGetErrorString(cuda_error));
        return;
    }

    cuda_error = hipMalloc((void**)&d_sobel, width * height * sizeof(int));
    if (cuda_error != hipSuccess) {
        fprintf(stderr, "(hipMalloc d_sobel): %s\n", hipGetErrorString(cuda_error));
        hipFree(d_image);  
        return;
    }

    cuda_error = hipMemcpy(d_image, image, width * height * sizeof(int), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        fprintf(stderr, "(hipMemcpy host to device): %s\n", hipGetErrorString(cuda_error));
        hipFree(d_image);  
        hipFree(d_sobel);  
        return;
    }

    dim3 block_size(32, 32);
    // gridSize must be s.t. we can fit the whole image
    dim3 grid_size((width + block_size.x - 1) / block_size.x + 1, (height + block_size.y - 1) / block_size.y + 1);
    sobel_filter_kernel<<<grid_size, block_size>>>(d_image, d_sobel, width, height);
    hipFree(d_image);
    cuda_error = hipMemcpy(image, d_sobel, width*height * sizeof(int), hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess) {
        fprintf(stderr, "(hipMemcpy device to host): %s\n", hipGetErrorString(cuda_error));
        hipFree(d_image);  
        hipFree(d_sobel);  
        return;
    }
    hipFree(d_sobel);
}

